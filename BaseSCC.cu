#include "hip/hip_runtime.h"
/*
   *REALLY naive Implementation of FWBW

   */

#include <cstdlib>
#include <cstdio>
#include <stack>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <unistd.h>
#include <ctype.h>

typedef struct Node {
	unsigned int order;
	unsigned int out_degree;
	unsigned int in_degree;
	unsigned int * successors;
	unsigned int * predecessors;
	int scc;
	unsigned int fw_reachable;
	unsigned int bw_reachable;
	int subgraph;
} Node;

__constant__ unsigned int d_num_nodes;

__global__ void bfs(Node * graph, unsigned int * finished, unsigned int * mode, int * subgraph){

	if (threadIdx.x == 0) {atomicCAS(finished, 0, 1);}
	//syncthreads
	__threadfence();

	unsigned int old;

	unsigned int v = (blockIdx.x * blockDim.x) + threadIdx.x;

	Node curr_node, neighbor;
	
	for (unsigned int i = v; i < d_num_nodes; i++) {
		
		//get the current node
		curr_node = graph[i];

		//if the current node is in the subgraph
		if (curr_node.subgraph == *subgraph) {
			//if mode == 0, it's a forward search
			if (*mode == 0) {
				//for each of it's neighbors
				for(int i = 0; i < curr_node.out_degree; i++){
					//if the current node is forward reachable and the neighbor is in the subgraph
					neighbor = graph[curr_node.successors[i]];
					if (curr_node.fw_reachable && neighbor.subgraph == *subgraph) {
						//tell it's neighbors they are forward reachable
						old = atomicCAS(&(graph[curr_node.successors[i]].fw_reachable), 0, 1);
						//if something changed, we aren't at stasis
						if (!old) {atomicCAS(finished, 1, 0);}
					}
				}
			}
			//mode is 1, and it's a backward search on the transpose graph
			else {
				for(int i = 0; i < curr_node.in_degree; i++){
					if (curr_node.bw_reachable && neighbor.subgraph == *subgraph) {
						//tell the neighbors they are backwards reachable
						old = atomicCAS(&(graph[curr_node.predecessors[i]].bw_reachable), 0, 1);
						//if something changed, we aren't at stasis
						if (!old) {atomicCAS(finished, 1, 0);}
					}
				}
			}
		}
	}
}

__global__ void reset_bwfw_reachability(Node * graph){
	//reset node.fw_reachable & node.bw_reachable	
	 
	unsigned int v = (blockIdx.x * blockDim.x) + threadIdx.x;
	Node curr_node;
	
	for (unsigned int i = v; i < d_num_nodes; i++){	
		//get current node
		curr_node = graph[i];
		//reset it
		curr_node.fw_reachable = 0;
		curr_node.bw_reachable = 0;
	}
}

__global__ void trim_kernel(Node * graph, unsigned int * finished, int * subgraph) {

	if (threadIdx.x == 0) {atomicCAS(finished, 0, 1);}

	unsigned int v = (blockIdx.x * blockDim.x) + threadIdx.x;
	Node curr_node, neighbor;
	
	for (unsigned int i = v; i < d_num_nodes; i++){
		
		//get the current node
		curr_node = graph[i];
		
		//if the current node is in the subgraph
		if (curr_node.subgraph == *subgraph) {
			//visit each neighbor in the subgraph, telling them they are reachable
			for(int i = 0; i < curr_node.out_degree; i++){
				neighbor = graph[curr_node.successors[i]];
				if (neighbor.subgraph == *subgraph) {
					atomicCAS(&(graph[curr_node.successors[i]].fw_reachable), 0, 1);
				}
			}
		}
	}
		
	//sync threads, may need threadfence?
	__threadfence();

	
	for (unsigned int i = v; i < d_num_nodes; i++){
		//if you are not reachable or do not reach
		if (curr_node.fw_reachable == 0 || curr_node.out_degree == 0) {
			//then you are trimmable
			curr_node.scc = curr_node.order;
			curr_node.subgraph = -1;
			atomicCAS(finished, 1, 0);
		}
	}
} 

__global__ void assign_scc(Node * graph, int * scc) {
	//We want to find the smallest order of a node in the subgraph both reachable from and by the pivot
	//That will be our SCC label
	
	//This implementation is just an atomic min; a reduction could yeild much better throughput

	unsigned int v = (blockIdx.x * blockDim.x) + threadIdx.x;
	Node curr_node;
	
	for (unsigned int i = v; i < d_num_nodes; i++){
		
		//get the current node
		curr_node = graph[i];

		if (curr_node.fw_reachable && curr_node.bw_reachable) {atomicMin(scc, curr_node.order);}
		
	}
	
	//make sure all atomic writes are done
	__threadfence();
	
	//scc should be the lowest now
	curr_node.scc = *scc;
	curr_node.subgraph = -1;
	
}

__global__ void ancestor_partition(Node * graph, int * subgraph, unsigned int * empty){
	
	//We need to assign the anscestors who are not in the scc to a new subgraph
	unsigned int v = (blockIdx.x * blockDim.x) + threadIdx.x;
	Node curr_node;

	if (threadIdx.x == 0) {atomicCAS(empty, 0, 1);}
	
	__threadfence();
	
	for (unsigned int i = v; i < d_num_nodes; i++){
		
		//get the current node
		curr_node = graph[i];

		if (curr_node.bw_reachable && !curr_node.fw_reachable){curr_node.subgraph = *subgraph; atomicCAS(empty, 1, 0);}
	}
}

__global__ void descendent_partition(Node * graph, int * subgraph, unsigned int * empty){

	//We need to assign the descendents who are not in the scc to a new subgraph
	unsigned int v = (blockIdx.x * blockDim.x) + threadIdx.x;
	Node curr_node;

	if (threadIdx.x == 0) {atomicCAS(empty, 0, 1);}
	
	__threadfence();
	
	for (unsigned int i = v; i < d_num_nodes; i++){
		
		//get the current node
		curr_node = graph[i];

		if (!curr_node.bw_reachable && curr_node.fw_reachable){curr_node.subgraph = *subgraph; atomicCAS(empty, 1, 0);}
	}
}

__global__ void remainder_partition(Node * graph, int * subgraph, unsigned int * empty){
	//We need to assign the descendents who are not in the scc to a new subgraph
	unsigned int v = (blockIdx.x * blockDim.x) + threadIdx.x;
	Node curr_node;

	if (threadIdx.x == 0) {atomicCAS(empty, 0, 1);}
	
	__threadfence();
	
	for (unsigned int i = v; i < d_num_nodes; i++){
		
		//get the current node
		curr_node = graph[i];

		if (!curr_node.bw_reachable && !curr_node.fw_reachable){curr_node.subgraph = *subgraph; atomicCAS(empty, 1, 0);}
	}
}

__global__ void pivot(Node * graph, int * subgraph, unsigned int * pivot){
	//here we just assign the lowest order node in a subgraph to be the pivot
	//this is suboptimal

	//pivot should be initialized to num_nodes

	unsigned int v = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int votes;
	unsigned int lowest;
	unsigned int lane_id = threadIdx.x % 32;
	Node curr_node;

	for (unsigned int i = v; i < d_num_nodes; i++){
		//we don't want to hammer away with atomics, so we reduce the number of
		//threads using bandwidth

		//this can be further optimized		

		//get the current node
		curr_node = graph[i];
		//see how many nodes in the warp are in the subgraph
		votes = __ballot(curr_node.subgraph == *subgraph);
		//find the first node, and see if it's this node
		lowest = __ffs(votes) - 1;
		if (lane_id == lowest) atomicMin(pivot, curr_node.order);
	}

	__threadfence();
	//now that pivot should be finished, we need to initialize the corresponding node
	for (unsigned int i = v; i < d_num_nodes; i++){
		curr_node = graph[i];
		if (lane_id == lowest && *pivot == curr_node.order) {
			curr_node.fw_reachable = 1;
			curr_node.bw_reachable = 1;
		}
	}
}
		

int main(int argc, char ** argv){

	printf("Basic FWBW SCC v1.0\n");
	int option;
	bool output = false;
	char * out_file, * in_file;
	bool trim = false;
	unsigned int blocks = 0;
	unsigned int threads = 0;		
	
	while ((option = getopt(argc, argv, "o:tb:x:")) != -1) {
		switch (option) {
			case 'o':
				output = true;
				out_file = optarg;
				break;
			case 't':
				trim = true;
				break;
			case 'b':
				blocks = (unsigned int) atoi(optarg);
				break;
			case 'x':
				threads = (unsigned int) atoi(optarg);
				break;
			case '?':
				if (optopt == 'o' || optopt == 'b' || optopt == 'x') {fprintf(stderr, "Option -%c requires an argument\n", optopt); exit(-1);}
				else if (isprint(optopt)) {fprintf(stderr, "Unknown option -%c\n", optopt); exit(-1);}
				else {fprintf(stderr, "Unknown option \\x%x\n", optopt); exit(-1);}
			default:
				exit(-1);
		}
	}
	
	if (optind >= argc || optind < argc - 1) {fprintf(stderr, "Input Graph File Required\n"); exit(-1);}
	
	in_file = argv[optind];

	FILE * fp = fopen(in_file, "r");
	if (!fp) {fprintf(stderr, "Error Reading Graph File"); exit(-1);}
	
	char * line = NULL;
	size_t len = 0;
	ssize_t read;
	
	Node * graph = NULL;
	
	//read the graph from the file
	//in this implementation, the graph representation is AOS
	int num_nodes, num_edges;
	int source, dest;
	
	//initialize based on header in graph file and get the degree of each node
	while((read=getline(&line, &len, fp)) != -1){
		//ignore header lines
		if (line[0] == 'p'){
			//this line has the total number of nodes and edges, we can make an empty graph
			sscanf(line, "p sp %d %d", &num_nodes, &num_edges);
			graph = (Node *) malloc(sizeof(Node) * num_nodes);
			for (int i = 0; i < num_nodes; i++) {
				graph[i].out_degree = 0;
				graph[i].successors = NULL;
				}
		}
		else if (line[0] == 'a'){
			//now we know how much memory to allocate for each node
			sscanf(line, "a %d %d %*d", &source, &dest);
			graph[source - 1].out_degree++;
			graph[dest - 1].in_degree++;
			}
	}	
	if (graph == NULL) {fprintf(stderr, "Error Reading Graph File, No Graph Allocated"); exit(-1);}
	rewind(fp);
	for (int i = 0; i < num_nodes; i++){
		//allocate the memory for each node
		graph[i].successors = (unsigned int *) malloc(sizeof(unsigned int) * graph[i].out_degree);
		graph[i].predecessors = (unsigned int *) malloc(sizeof(unsigned int) * graph[i].in_degree);
	}
	int s = 0;
	int c = 0;
	unsigned int * predec_tracker = (unsigned int *) calloc(num_nodes, sizeof(unsigned int));
	//record edges in the graph structure
	//the format is sorted by source node
	while((read=getline(&line, &len, fp)) != -1){
		if (line[0] == 'a'){
			sscanf(line, "a %d %d %*d", &source, &dest);
			//record the transverse edge
			graph[dest - 1].predecessors[predec_tracker[dest - 1]] = (source - 1);
			predec_tracker[dest - 1]++;
			//record the edge, done this way we don't need another large array
			if (source - 1 == s){
				graph[s].successors[c] = (dest - 1);
				c++;
			}
			else {
				s = source - 1;
				c = 0;
				graph[s].successors[c] = (dest - 1);
				c++;
			}
		}
	}

	fclose(fp);
	free(line);
	free(predec_tracker);

	//Do some preprocessing and error checking here on the incoming graph
	for (unsigned int i = 0; i < num_nodes; i++){
		graph[i].order = i;
		graph[i].scc = -1;
		graph[i].subgraph = 0;
	}
			
	//if blocks and threads aren't pre-assigned, assign them	
	if (threads == 0) {threads = 512;}
	if (blocks == 0) {blocks = (num_nodes + (threads - 1)) / threads;}

	//set up the stack
	std::stack<unsigned int> subgraphs;
	subgraphs.push(0);
		
	//We want to time both the kernel and the all of the relevant processes
	timeval allstart, allend, kstart, kend;
	gettimeofday(&allstart, NULL);

	//Allocate the necessary memory on the device, and copy over any initial data
	Node * d_graph;
	unsigned int h_finished, h_mode, h_subgraph, h_empty;
	unsigned int * d_finished, * d_mode, * d_empty, * d_pivot;
	int * d_subgraph, * d_scc;
	
	if (hipSuccess != hipMalloc((void **) &d_graph, sizeof(Node) * num_nodes)) {fprintf(stderr, "Couldn't allocate d_graph\n"); exit(-1);}
	if (hipSuccess != hipMalloc((void **) &d_finished, sizeof(unsigned int))) {fprintf(stderr, "Couldn't allocate d_finished\n"); exit(-1);}
	if (hipSuccess != hipMalloc((void **) &d_mode, sizeof(unsigned int))) {fprintf(stderr, "Couldn't allocate d_mode\n"); exit(-1);}
	if (hipSuccess != hipMalloc((void **) &d_subgraph, sizeof(int))) {fprintf(stderr, "Couldn't allocate d_subgraph\n"); exit(-1);}
	if (hipSuccess != hipMalloc((void **) &d_scc, sizeof(int))) {fprintf(stderr, "Couldn't allocate d_scc\n"); exit(-1);}
	if (hipSuccess != hipMalloc((void **) &d_empty, sizeof(unsigned int))) {fprintf(stderr, "Couldn't allocate d_empty\n"); exit(-1);}
	if (hipSuccess != hipMalloc((void **) &d_pivot, sizeof(unsigned int))) {fprintf(stderr, "Couldn't allocate d_pivot\n"); exit(-1);}

	hipMemcpyToSymbol(HIP_SYMBOL(d_num_nodes), &num_nodes, sizeof(unsigned int));
	
	//we need to copy over the graph struct, and allocate/copy the predecessor and successor arrays
	if (hipSuccess != hipMemcpy(d_graph, graph, sizeof(Node) * num_nodes, hipMemcpyHostToDevice)) {fprintf(stderr, "Couldn't copy graph to device\n"); exit(-1);}
	unsigned int * d_successors, * d_predecessors;
	for (int i = 0; i < num_nodes; i++){
		if (hipSuccess != hipMalloc((void **) &d_successors, graph[i].out_degree * sizeof(unsigned int))) {fprintf(stderr, "Couldn't allocate a successors array for node %d\n", i); exit(-1);}
		if (hipSuccess != hipMalloc((void **) &d_predecessors, graph[i].in_degree * sizeof(unsigned int))) {fprintf(stderr, "Couldn't allocate a predecessors array for node %d\n", i); exit(-1);}
		if (hipSuccess != hipMemcpy(d_successors, graph[i].successors, graph[i].out_degree * sizeof(unsigned int), hipMemcpyHostToDevice)) {fprintf(stderr, "Couldn't copy successors array for node %d\n", i); exit(-1);}
		if (hipSuccess != hipMemcpy(d_predecessors, graph[i].predecessors, graph[i].in_degree * sizeof(unsigned int), hipMemcpyHostToDevice)) {fprintf(stderr, "Couldn't copy predeccessors array for node %d\n", i); exit(-1);}
		if (hipSuccess != hipMemcpy(&(d_graph->successors), &d_successors, sizeof(unsigned int *), hipMemcpyHostToDevice)) {fprintf(stderr, "Couldn't copy successors pointer for node %d\n", i); exit(-1);}
		if (hipSuccess != hipMemcpy(&(d_graph->predecessors), &d_predecessors, sizeof(unsigned int *), hipMemcpyHostToDevice)) {fprintf(stderr, "Couldn't copy predecessors pointer for node %d\n", i); exit(-1);}
	}

	//Start the kernel timer
	gettimeofday(&kstart, NULL);

	//while there's still work to do (subgraphs on the stack)
	while (!subgraphs.empty()){
	
		//take a subgraph off the stack	
		h_subgraph = subgraphs.top();
		subgraphs.pop();
		if (hipSuccess != hipMemcpy(d_subgraph, &h_subgraph, sizeof(unsigned int), hipMemcpyHostToDevice)) {fprintf(stderr, "Couldn't copy d_subgraph to device\n"); exit(-1);}

		//reset it's forward and backward reachable attributes
		reset_bwfw_reachability<<<blocks, threads>>>(d_graph);

		//if trim, then trim till you can trim no more
		if (trim){
			h_finished = 0;
			if (hipSuccess != hipMemcpy(d_finished, &h_finished, sizeof(unsigned int), hipMemcpyHostToDevice)) {fprintf(stderr, "Couldn't copy d_finished to device\n"); exit(-1);}
			while (!h_finished){
				//trim
				trim_kernel<<<blocks, threads>>>(d_graph, d_finished, d_subgraph);
				//reset the reachability
				reset_bwfw_reachability<<<blocks, threads>>>(d_graph);
				//check if finished
				if (hipSuccess != hipMemcpy(&h_finished, d_finished, sizeof(unsigned int), hipMemcpyDeviceToHost)) {fprintf(stderr, "Couldn't copy d_finished to host\n"); exit(-1);}
			}
		}
						
		//choose a pivot
		pivot<<<blocks, threads>>>(d_graph, d_subgraph, d_pivot);

		//do a forwards reachability search
		h_finished = 0;
		h_mode = 0;
		if (hipSuccess != hipMemcpy(d_finished, &h_finished, sizeof(unsigned int), hipMemcpyHostToDevice)) {fprintf(stderr, "Couldn't copy d_finished to device\n"); exit(-1);}
		if (hipSuccess != hipMemcpy(d_mode, &h_mode, sizeof(unsigned int), hipMemcpyHostToDevice)) {fprintf(stderr, "Couldn't copy d_mode to device\n"); exit(-1);}
		while (!h_finished){
			bfs<<<blocks, threads>>>(d_graph, d_finished, d_mode, d_subgraph);
			if (hipSuccess != hipMemcpy(&h_finished, d_finished, sizeof(unsigned int), hipMemcpyDeviceToHost)) {fprintf(stderr, "Couldn't copy d_finished to host\n"); exit(-1);}
		}

		//do a backwards reachability search
		h_finished = 0;
		h_mode = 1;
		if (hipSuccess != hipMemcpy(d_finished, &h_finished, sizeof(unsigned int), hipMemcpyHostToDevice)) {fprintf(stderr, "Couldn't copy d_finished to device\n"); exit(-1);}
		if (hipSuccess != hipMemcpy(d_mode, &h_mode, sizeof(unsigned int), hipMemcpyHostToDevice)) {fprintf(stderr, "Couldn't copy d_mode to device\n"); exit(-1);}
		while (!h_finished){
			bfs<<<blocks, threads>>>(d_graph, d_finished, d_mode, d_subgraph);
			if (hipSuccess != hipMemcpy(&h_finished, d_finished, sizeof(unsigned int), hipMemcpyDeviceToHost)) {fprintf(stderr, "Couldn't copy d_finished to host\n"); exit(-1);}
		}

		//Add the nodes in the union of the FW and BW reachable sets to an SCC
		assign_scc<<<blocks, threads>>>(d_graph, d_scc);

		//Add the FW reachable nodes not in the SCC to the subgraph list
		h_subgraph++;
		if (hipSuccess != hipMemcpy(d_subgraph, &h_subgraph, sizeof(unsigned int), hipMemcpyHostToDevice)) {fprintf(stderr, "Couldn't copy d_finished to device\n"); exit(-1);}
		descendent_partition<<<blocks, threads>>>(d_graph, d_subgraph, d_empty);
		if (hipSuccess != hipMemcpy(&h_empty, d_empty, sizeof(unsigned int), hipMemcpyDeviceToHost)) {fprintf(stderr, "Couldn't copy d_finished to host\n"); exit(-1);}
		if (!h_empty) {subgraphs.push(h_subgraph);}

		//Add the BW reachable nodes not in the SCC to the subgraph list
		h_subgraph++;
		if (hipSuccess != hipMemcpy(d_subgraph, &h_subgraph, sizeof(unsigned int), hipMemcpyHostToDevice)) {fprintf(stderr, "Couldn't copy d_finished to device\n"); exit(-1);}
		ancestor_partition<<<blocks, threads>>>(d_graph, d_subgraph, d_empty);
		if (hipSuccess != hipMemcpy(&h_empty, d_empty, sizeof(unsigned int), hipMemcpyDeviceToHost)) {fprintf(stderr, "Couldn't copy d_finished to host\n"); exit(-1);}
		if (!h_empty) {subgraphs.push(h_subgraph);}

		//Add the nodes not reachable from the pivot to to the subgraph list
		h_subgraph++;
		if (hipSuccess != hipMemcpy(d_subgraph, &h_subgraph, sizeof(unsigned int), hipMemcpyHostToDevice)) {fprintf(stderr, "Couldn't copy d_finished to device\n"); exit(-1);}
		remainder_partition<<<blocks, threads>>>(d_graph, d_subgraph, d_empty);
		if (hipSuccess != hipMemcpy(&h_empty, d_empty, sizeof(unsigned int), hipMemcpyDeviceToHost)) {fprintf(stderr, "Couldn't copy d_finished to host\n"); exit(-1);}
		if (!h_empty) {subgraphs.push(h_subgraph);}
	}
	//end the timers
	gettimeofday(&kend, NULL);
	gettimeofday(&allend, NULL);

	//Print the Times
	double all_runtime = allend.tv_sec + allend.tv_usec / 1000000.0 - allstart.tv_sec - allstart.tv_usec / 1000000.0;
	double kruntime = kend.tv_sec + kend.tv_usec / 1000000.0 - kstart.tv_sec - kstart.tv_usec / 1000000.0;

	for (int i = 0; i < num_nodes; i++) {
		free(graph[i].successors);
		free(graph[i].predecessors);

	//Print the Results, if necessary
	if (hipSuccess != hipMemcpy(graph, d_graph, sizeof(Node) * num_nodes, hipMemcpyDeviceToHost)) {fprintf(stderr, "Couldn't copy graph to host\n"); exit(-1);}
	
	if (output == true) {
		fp = fopen(out_file, "w+");
		int scc = 0;	
		int count = 0;
		bool first;
		while (count < num_nodes) {
			first = true;
			for (int i = 0; i < num_nodes; i++) {
				if (graph[i].scc == scc) {
					count++;
					if (first) {
						fprintf(fp, "\nSCC %d: ", scc);
						first = false;
					}
					fprintf(fp, "%d ", graph[i].order);
				}

			}
			scc++;
		}
		fprintf(fp, "\n");
		fclose(fp);
	}

	//Free the memory
	free(graph);
	hipFree(d_finished);
	hipFree(d_mode);
	hipFree(d_scc);
	hipFree(d_pivot);
	hipFree(d_empty);
	hipFree(d_subgraph);
	for (int i = 0; i < num_nodes; i++) {
		hipFree(d_graph->successors);
		hipFree(d_graph->predecessors);
	}
	hipFree(d_graph);

	return 0;

}
	
	
