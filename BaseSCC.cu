/*
   Implementation of ColoringSCC

   */

#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <sys/time.h>

struct i_Node {
	int in_degree;
	int * in_neghbors;
};

int main(int argc, char*argv[]){

	printf("Coloring SCC v1.0\n");
	
	//check command line for arguments; we need a graph for input
	if (argc != 2 and argc != 3) {fprintf(stderr, "usage: %s <graph_file> <output_file>(optional)\n", argv[0]); exit(-1);}
	
	//read the graph from the file, creating an adjacency matrix as representation
	//Graph file should be formatted as a multiline adjancency list, and the first
	//line should be the number of nodes in the graph
	int num_nodes;

	fp = fopen(argv[1], "r");
	if (!fp) {fprintf(stderr, "Error Reading Graph File"); exit(-1);}

	fscanf(fp, "%d", &num_nodes); //first line should be the number of nodes
	
	Node * h_graph = (Node *) malloc(sizeof(Node) * num_nodes);

	int node, degree, predecessor;
	for (int i = 0; i < num_nodes; i++){
		fscanf(fp, "%d %d", &start, &degree);
		h_graph[start].in_degree = degree
		h_graph[start].in_neighbors = (int *) malloc(sizeof(int) * degree);
		for (int i = 0; i < degree; i++) {fscanf(fp, "%d", &h_graph[start].in_neighbors[i]);}
	}

	//Do some preprocessing and error checking here on the incoming graph
	
	
	//We want to time both the kernel and the all of the relevant processes
	timeval allstart, allend, kstart, kend;
	gettimeofday(&allstart, NULL);

	//Allocate the necessary memory on the device


	//Start the kernel timer
	gettimeofday(&kstart, NULL);

	//Call the kernel

	//end the timers
	gettimeofday(&kend, NULL);
	gettimeofday(&allend, NULL);

	//Print the Times
	double all_runtime = allend.tv_sec + allend.tv_usec / 1000000.0 - allstart.tv_sec - allstart.tv_usec / 1000000.0;
	double kruntime = kend.tv_sec + kend.tv_usec / 1000000.0 - kstart.tv_sec - kstart.tv_usec / 1000000.0;

	//Verify the Results
	

	//Free the memory
	for (int i = 0; i < num_nodes; i++) {
		free(h_graph[i].in_neighbors);
	free(h_graph)

	return 0;

}
	
	
