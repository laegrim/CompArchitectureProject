/*
   Implementation of ColoringSCC

   */

#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <sys/time.h>

int main(int argc, char*argv[]){

	printf("Coloring SCC v1.0\n");
	
	//check command line for arguments; we need a graph for input
	if (argc != 2) {fprintf(stderr, "usage: %s graph_file\n", argv[0]); exit(-1);}
	
	//Do some preprocessing and error checking here on the incoming graph
	
	
	//We want to time both the kernel and the all of the relevant processes
	timeval allstart, allend, kstart, kend;
	gettimeofday(&allstart, NULL);

	//Allocate the necessary memory on the device


	//Start the kernel timer
	gettimeofday(&kstart, NULL);

	//Call the kernel

	//end the timers
	gettimeofday(&kend, NULL);
	gettimeofday(&allend, NULL);

	//Print the Times


	//Verify the Results


	//Free the memory


	return 0;

}
	
	
