#include "hip/hip_runtime.h"
#ifndef _KERNELS_H_
#define _KERNELS_H_

struct Node {

}

__global__ void FWBW(Node * d_Graph, Node &pivot) {

	//If d_Graph

		//Get FW Set

		//Get BW Set

		//Find Intersection

		//Reorder Global Data

		//Call Recursively on Predecessors not in SCC
	
		//Call Recursively on Sucessors not in SCC

		//Call Recursively on Remainder

}

__global__ void FWBWTrim(Node * d_Graph, Node &pivot) {
	
	//if d_Graph

		//Trim Graph		
		
		//Get FW Set

		//Get BW Set

		//Find Intersection

		//Reorder Global Data

		//Call Recursively on Predecessors not in SCC

		//Call Recursively on Successors not in SCC

		//Call Recursively on Remainder

}

__global__ void Trim() {

	//Remove nodes with either no predecessor or no anscestor

	//Reorder Global Data

	//Call Recursively
}

//__global__ void ColorProp() {

//}
